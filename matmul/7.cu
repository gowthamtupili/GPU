
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
// #include <cstdio>
// #include <cstdlib>
// #include <cublas_v2.h>
// #include <cuda_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

using std::cout;
using std::vector;

typedef unsigned int uint32;

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

const uint32 BK = 8;
const uint32 TM = 8;
const uint32 TN = 8;
const uint32 BM = 128;
const uint32 BN = 128;


void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
  }


__global__ void myGEMM7(int M, int N, int K, float *A,
                               float *B, float *C) {
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;

  // BN/TN are the number of threads to span a column
  const int threadCol = threadIdx.x % (BN / TN);
  const int threadRow = threadIdx.x / (BN / TN);

  // allocate space for the current blocktile in smem
  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * (BN+5)];

  // Move blocktile to beginning of A's row and B's column
  A += cRow * BM * K;
  B += cCol * BN;
  C += cRow * BM * N + cCol * BN;

  // calculating the indices that this thread will load into SMEM
  // we'll load 128bit / 32bit = 4 elements per thread at each step
  const uint innerRowA = threadIdx.x / (BK / 4);
  const uint innerColA = threadIdx.x % (BK / 4);
  const uint innerRowB = threadIdx.x / (BN / 4);
  const uint innerColB = threadIdx.x % (BN / 4);

  // allocate thread-local cache for results in registerfile
  float threadResults[TM * TN] = {0.0};
  float regM[TM] = {0.0};
  float regN[TN] = {0.0};

  // outer-most loop over block tiles
  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    // populate the SMEM caches
    // transpose A while loading it
    float4 tmp =
        reinterpret_cast<float4 *>(&A[innerRowA * K + innerColA * 4])[0];
    As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
    As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
    As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
    As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

    reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] =
        reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];
    __syncthreads();

    // advance blocktile
    A += BK;     // move BK columns to right
    B += BK * N; // move BK rows down

    // calculate per-thread results
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
      // block into registers
      for (uint i = 0; i < TM; ++i) {
        regM[i] = As[dotIdx * BM + threadRow * TM + i];
      }
      for (uint i = 0; i < TN; ++i) {
        regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
      }
      for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
          threadResults[resIdxM * TN + resIdxN] +=
              regM[resIdxM] * regN[resIdxN];
        }
      }
    }
    __syncthreads();
  }

  // write out the results
  for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
    for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
      // load C vector into registers
      float4 tmp = reinterpret_cast<float4 *>(
          &C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0];
      // perform GEMM update in reg
      tmp.x = threadResults[resIdxM * TN + resIdxN] + tmp.x;
      tmp.y = threadResults[resIdxM * TN + resIdxN + 1] + tmp.y;
      tmp.z = threadResults[resIdxM * TN + resIdxN + 2] + tmp.z;
      tmp.w = threadResults[resIdxM * TN + resIdxN + 3] + tmp.w;
      // write back
      reinterpret_cast<float4 *>(
          &C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0] =
          tmp;
    }
  }
}



// Check result on the CPU
void verify_result(vector<float> &a, vector<float> &b, vector<float> &c, int M, int N, int K) {
    auto start = std::chrono::high_resolution_clock::now();
    // cout << "Matrix C (CPU) \n";
    for (int i = 0; i < M; i++) {
      for (int j = 0; j < K; j++) {
        // For every element in the row-column pair
        int tmp = 0;
        for (int k = 0; k < N; k++) {
          // Accumulate the partial results
          tmp += a[i * N + k] * b[k * K + j];
        }
        // Check against the CPU result
        // cout << tmp << " ";
        // if(tmp = c[i * K + j]) {
        //     cout << i << " " << j << "\n";
        // }
        assert(tmp == c[i * K + j]);
      }
    //   cout << "\n";
    }
    // Calculate elapsed time
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;
    std::cout << "CPU Mat Mul Time: " << duration.count() << " ms\n";
}


int main () {

    // Matrix size of 1024 x 1024;
    int M = 1 << 10;
    int N = 1 << 10;
    int K = 1 << 10;

    size_t bytes1 = M * (K) * sizeof(float);
    size_t bytes2 = K * (N) * sizeof(float);
    size_t bytes3 = M * (N) * sizeof(float);

    vector<float> h_a(M * K);
    vector<float> h_b(K * N);
    vector<float> h_c(M * N);


    // Initialize matrices
    for(int i=0;i<h_a.size();i++) {
        h_a[i] = rand() % 100;
        // h_a[i] = i *2;
        // h_a[i] = 1.0f;
    }
    for(int i=0;i<h_b.size();i++) {
        h_b[i] = rand() % 100;
        // h_b[i] = i * 10;
        // h_b[i] = 1.0f;
    }
//   cout << "Matrix A: " << M << "x" << N << "\n";
//   for(int i = 0; i < M; i++) {
//     for(int j = 0; j < N; j++) {
//       cout << h_a[i*N + j] << " ";
//     }
//     cout << "\n";
//   }

//   cout << "Matrix B: " << N << "x" << K << "\n";
//   for(int i = 0; i < N; i++) {
//     for(int j = 0; j < K; j++) {
//       cout << h_b[i*K + j] << " ";
//     }
//     cout << "\n";
//   }

    // Variable to measure time.
    hipEvent_t start, stop, kernelStart, kernelStop;
    float time = 0, kernelTime = 0; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    
    hipEventRecord(start);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes1);
    hipMalloc(&d_b, bytes2);
    hipMalloc(&d_c, bytes3);

    // Copy data to the device
    hipMemcpy(d_a, h_a.data(), bytes1, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes2, hipMemcpyHostToDevice);
    checkCudaError(hipGetLastError(), "Memory Transfer failed");
    // Use dim3 structs for block  and grid dimensions

    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

	hipEventRecord(kernelStart);

    myGEMM7 <<<gridDim, blockDim>>>(M, N, K, d_a, d_b, d_c);

    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);

    // cudaDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Kernel launch failed");
  
  
    // Copy back to the host
    hipMemcpy(h_c.data(), d_c, bytes3, hipMemcpyDeviceToHost);
    // cout << "Matrix C(GPU): " << M << "x" << N << "\n";
    // for(int i = 0; i < M; i++) {
    //     for(int j = 0; j < N; j++) {
    //     cout << h_c[i*K + j] << " ";
    //     }
    //     cout << "\n";
    // }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
  
    hipEventElapsedTime(&time, start, stop);
    hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
    std::cout << "GPU Kernel time: " << kernelTime << " ms" << std::endl;
    std::cout << "GPU Total time (H2D + Kernel + D2H): " << time << " ms" << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    verify_result(h_a, h_b, h_c, M, K, N);

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}