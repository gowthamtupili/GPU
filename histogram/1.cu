// Naive Implementation


#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <vector>
#include <chrono>


using std::cout;
using std::vector;


#define THREADS 256

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
  


__global__ void histogram(char* input, int* output, int N) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < N) {
        atomicAdd(&output[input[tid] - 'A'], 1);
    }
};



void verify_results(char* input, int* output, int N) {
    auto start = std::chrono::high_resolution_clock::now();
    int temp[26] = {0};
    for(int i=0;i<N;i++) {
        temp[input[i] - 'A']++;
    }
    // Calculate elapsed time
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;
    std::cout << "CPU Time: " << duration.count() << " ms\n";
    for(int i=0;i<26;i++) {
        // cout << temp[i] << " ";
        assert(temp[i] == output[i]);
    }
}

int main () {
    int N = 1 << 24;
    int bins = 26;
    vector<char> input(N);
    vector<int> result(bins);
    size_t bytes1 = N * sizeof(char);
    size_t bytes2 = 26 * sizeof(int);

    for(int i=0;i<N;i++) {
        input[i] = 65 + rand() % 26;
    }

    // Variable to measure time.
    hipEvent_t start, stop, kernelStart, kernelStop;
    float time = 0, kernelTime = 0; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    
    hipEventRecord(start);

    char* d_input;
    int* d_result;
    hipMalloc(&d_input, bytes1);
    hipMalloc(&d_result, bytes2);
    hipMemcpy(d_input, input.data(), bytes1, hipMemcpyHostToDevice);


    dim3 threads(THREADS);
    dim3 blocks(N/THREADS);

	hipEventRecord(kernelStart);

    histogram<<<blocks, threads>>> (d_input, d_result, N);

    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Kernel launch failed");
  
    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);

    hipMemcpy(result.data(), d_result, bytes2, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
  
    hipEventElapsedTime(&time, start, stop);
    hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
    std::cout << "GPU Kernel time: " << kernelTime << " ms" << std::endl;
    std::cout << "GPU Total time (H2D + Kernel + D2H): " << time << " ms" << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    // for(auto i:result) cout << i << " ";
    // cout << "\n";
    verify_results(input.data(), result.data(), N);
    cout << "Completed Successfully\n";
    return 0;

}
