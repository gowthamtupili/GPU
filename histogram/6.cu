// Using Shared Memory


#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <vector>
#include <chrono>


using std::cout;
using std::vector;


#define THREADS 256
#define WPT 8

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
  


__global__ void histogram_vectorized(char* input, int* output, int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;
    __shared__ unsigned int SM[26];

    // Initialize shared memory histogram
    if (tx < 26)
        SM[tx] = 0;
    __syncthreads();

    // Vectorized processing (4 chars per int)
    int* input4 = reinterpret_cast<int*>(input);
    int total_threads = gridDim.x * blockDim.x;
    int n_vec = N / 4;

    // ----------------------------------------------------------------------------------------------
    // unrolling the loop using pragma
    #pragma unroll
    // ----------------------------------------------------------------------------------------------


    for (int i = tid; i < n_vec; i += total_threads) {
        int val = input4[i];

        // Extract 4 chars from the int
        char c0 = (val >> 0)  & 0xFF;
        char c1 = (val >> 8)  & 0xFF;
        char c2 = (val >> 16) & 0xFF;
        char c3 = (val >> 24) & 0xFF;

        atomicAdd(&SM[c0 - 'A'], 1);
        atomicAdd(&SM[c1 - 'A'], 1);
        atomicAdd(&SM[c2 - 'A'], 1);
        atomicAdd(&SM[c3 - 'A'], 1);
    }

    // Handle remaining characters (tail)
    int tail_start = n_vec * 4;
    for (int i = tid + tail_start; i < N; i += total_threads) {
        char val = input[i];
        atomicAdd(&SM[val - 'A'], 1);
    }

    __syncthreads();

    // Global atomic update
    if (tx < 26)
        atomicAdd(&output[tx], SM[tx]);
}



void verify_results(char* input, int* output, int N) {
    auto start = std::chrono::high_resolution_clock::now();
    int temp[26] = {0};
    for(int i=0;i<N;i++) {
        temp[input[i] - 'A']++;
    }
    // Calculate elapsed time
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;
    std::cout << "CPU Time: " << duration.count() << " ms\n";
    for(int i=0;i<26;i++) {
        // cout << temp[i] << " ";
        assert(temp[i] == output[i]);
    }
}

int main () {
    int N = 1 << 24;
    int bins = 26;
    vector<char> input(N);
    vector<int> result(bins);
    size_t bytes1 = N * sizeof(char);
    size_t bytes2 = 26 * sizeof(int);

    for(int i=0;i<N;i++) {
        input[i] = 65 + rand() % 26;
    }

    // Variable to measure time.
    hipEvent_t start, stop, kernelStart, kernelStop;
    float time = 0, kernelTime = 0; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    
    hipEventRecord(start);

    char* d_input;
    int* d_result;
    hipMalloc(&d_input, bytes1);
    hipMalloc(&d_result, bytes2);
    hipMemcpy(d_input, input.data(), bytes1, hipMemcpyHostToDevice);


    dim3 threads(THREADS);
    // dim3 blocks(N/THREADS > 1 ? N/THREADS : 1);
    dim3 blocks((N / (4*WPT) + THREADS - 1) / THREADS);

	hipEventRecord(kernelStart);

    histogram_vectorized<<<blocks, threads>>> (d_input, d_result, N);

    // cudaDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Kernel launch failed");
  
    hipEventRecord(kernelStop);
    hipEventSynchronize(kernelStop);

    hipMemcpy(result.data(), d_result, bytes2, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
  
    hipEventElapsedTime(&time, start, stop);
    hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
    std::cout << "GPU Kernel time: " << kernelTime << " ms" << std::endl;
    std::cout << "GPU Total time (H2D + Kernel + D2H): " << time << " ms" << std::endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    // for(auto i:result) cout << i << " ";
    // cout << "\n";
    verify_results(input.data(), result.data(), N);
    cout << "Completed Successfully\n";
    return 0;

}